#include "hip/hip_runtime.h"
/**
 * @file      render.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Jiayi Chen, Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2023
 * @copyright Jiayi Chen, University of Pennsylvania
 */
#include <map>
#include <string>
#include <vector>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include "util/checkCUDAError.h"
#include "glm/gtc/quaternion.hpp"
#include "glm/gtx/transform.hpp"

#include "dataType.h"
#include "renderTool.h"
#include "renderFunc.h"
#include "render.h"


////////////////////////////////////////////////////////////////
///                      Render Pipeline                     ///
////////////////////////////////////////////////////////////////

void Render::render(const glm::mat4 & Model, const glm::mat4 & View, const glm::mat4 & Projection,
                    const int &beginW, const int &beginH, const int &bufferW, const int &bufferH, uchar4* const pbo) {
    // Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

    M = Model;
    V = View;
    P = Projection;

    // Vertex Process & primitive assembly
    {
        int curPrimitiveBeginId = 0; // change static to non-static
        dim3 numThreadsPerBlock(defaultThreadPerBlock);

        for(auto&& primitive:sceneInfo.mesh2PrimitivesMap)
        {
            for(auto&& p:primitive.second)
            {
                dim3 numBlocksForVertices((p.numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                dim3 numBlocksForIndices((p.numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

                _vertexTransform<<<numBlocksForVertices, numThreadsPerBlock>>>
                        (p.numVertices,
                         p,
                         M, V, P,
                         width,
                         height);
                checkCUDAError("Vertex Processing");
                _primitiveAssembly<<<numBlocksForIndices, numThreadsPerBlock>>>
                        (p.numIndices,
                         curPrimitiveBeginId,
                         dev_primitives,
                         p);
                checkCUDAError("Primitive Assembly");

                curPrimitiveBeginId += p.numPrimitives;
            }
        }

        checkCUDAError("Vertex Processing and Primitive Assembly");
    }

    {
        _clearTileBuffer<<<blockCount2d,1,1>>>(dev_tileBuffer,width,height,tileSize);
        checkCUDAError("Clear Tile Buffer");

        dim3 numThreadsPerBlock = defaultThreadPerBlock;
        dim3 numBlocks = (sceneInfo.numPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x;
        _generateTileBuffer<<<numBlocks,numThreadsPerBlock>>>
                (sceneInfo.numPrimitives,
                 dev_primitives,
                 dev_tileBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("Generate Tile Buffer");

        _rasterize<<<blockCount2d,blockSize2d,tileSize*tileSize*sizeof(Fragment)>>>
                (dev_primitives,
                 dev_tileBuffer,
                 dev_fragmentBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("Rasterize");

    }

    // Copy depthbuffer colors into framebuffer
    _fragmentShading<<<blockCount2d, blockSize2d>>>(dev_framebuffer,
                                                    dev_fragmentBuffer,
                                                    dev_lights,
                                                    sceneInfo.numLights,
                                                    overrideMaterial,
                                                    width,
                                                    height);
    checkCUDAError("Fragment Shader");

    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    _copyImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, dev_framebuffer, width, height,
                                                   beginW, beginH,bufferW, bufferH, true);
    checkCUDAError("Copy Render Result To Pbo");
}

void Render::inverseRender(const int &beginW, const int &beginH, const int &bufferW, const int &bufferH, uchar4* const pbo)
{
    _inverseFragmentShading<<<blockCount2d, blockSize2d>>>(dev_framebuffer,
                                                           dev_fragmentBuffer,
                                                           dev_lights,
                                                           sceneInfo.numLights,
                                                           width,
                                                           height);
    checkCUDAError("Inverse Fragment Shader");

    _inverseRasterize<<<blockCount2d,blockSize2d,tileSize*tileSize*sizeof(Fragment)>>>
            (dev_primitives,
             dev_tileBuffer,
             dev_fragmentBuffer,
             width,
             height,
             tileSize);
    checkCUDAError("Inverse Rasterize");

    {
        int curPrimitiveBeginId = 0;
        dim3 numThreadsPerBlock(defaultThreadPerBlock);

        for(auto&& primitive:sceneInfo.mesh2PrimitivesMap)
        {
            for(auto&& p:primitive.second)
            {
                dim3 numBlocksForVertices((p.numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                dim3 numBlocksForIndices((p.numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

                _inversePrimitiveAssembly<<<numBlocksForIndices, numThreadsPerBlock>>>
                        (p.numIndices,
                         curPrimitiveBeginId,
                         dev_primitives,
                         p);
                checkCUDAError("Inverse Primitive Assembly");

                _inverseVertexTransform<<<numBlocksForVertices, numThreadsPerBlock>>>
                        (p.numVertices,
                         p,
                         M, V, P,
                         width,
                         height);
                checkCUDAError("Inverse Vertex Processing");

                curPrimitiveBeginId += p.numPrimitives;
            }
        }

        checkCUDAError("Inverse Vertex Processing and Primitive Assembly");
    }

    _copyImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, dev_framebuffer, width, height,
                                                   beginW, beginH,bufferW, bufferH, true);
    checkCUDAError("Inverse Copy Image To PBO");
}

void Render::renderTex(int texIndex, const int &beginW, const int &beginH, const int &bufferW, const int &bufferH, uchar4* const pbo)
{
    // Assume show the first texture of the first primitive of the first mesh
    const Tex &tex = sceneInfo.mesh2PrimitivesMap.begin()->second.begin()->dev_tex[texIndex];
    _copyTexToPBO<<<blockCount2d, blockSize2d>>>(pbo, tex, width, height,
                                                 beginW, beginH,bufferW, bufferH, true);
}



////////////////////////////////////////////////////////////////
/// Functions that only be called when program start or exit ///
////////////////////////////////////////////////////////////////


/**
 * Called once at the end of the program to free CUDA memory.
 */
void Render::free() {

    // deconstruct primitives attribute/indices device buffer

    auto it(sceneInfo.mesh2PrimitivesMap.begin());
    auto itEnd(sceneInfo.mesh2PrimitivesMap.end());
    for (; it != itEnd; ++it) {
        for (auto p = it->second.begin(); p != it->second.end(); ++p) {
            hipFree(p->dev_indices);
            hipFree(p->dev_position);
            hipFree(p->dev_normal);
            hipFree(p->dev_uv);
            for(int i=0; i<maxTexNum; i++)
            {
                hipFree(p->dev_tex[i].data);
            }

            hipFree(p->dev_verticesOut);


            //TODO: release other attributes and materials
        }
    }

    ////////////

    hipFree(dev_primitives);
    dev_primitives = nullptr;

    hipFree(dev_fragmentBuffer);
    dev_fragmentBuffer = nullptr;

    hipFree(dev_tileBuffer);
    dev_tileBuffer = nullptr;

    hipFree(dev_framebuffer);
    dev_framebuffer = nullptr;

    hipFree(dev_lights);
    dev_lights = nullptr;

    checkCUDAError("render Free");
}

void Render::init(const tinygltf::Scene & scene, const std::vector<Light> &light, const int &w, const int &h) {

    // 0. Init some buffers that are not related to the scene
    {
        width = w;
        height = h;

        blockSize2d = {tileSize, tileSize, 1};
        blockCount2d ={(width - 1) / tileSize + 1,(height - 1) / tileSize + 1, 1};

        hipFree(dev_fragmentBuffer);
        hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
        hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
        hipFree(dev_tileBuffer);
        hipMalloc(&dev_tileBuffer,
                   ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
        hipMemset(dev_tileBuffer, 0,
                   ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
        hipFree(dev_framebuffer);
        hipMalloc(&dev_framebuffer, width * height * sizeof(glm::vec3));
        hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));


        sceneInfo.numLights = light.size();
        hipFree(dev_lights);
        hipMalloc(&dev_lights,sceneInfo.numLights * sizeof(Light));
        for(int i=0; i<sceneInfo.numLights; i++)
        {
            hipMemcpy(dev_lights+i, &(light[i]), sizeof(Light), hipMemcpyHostToDevice);
        }

        checkCUDAError("init");
    }

    int totalNumPrimitives = 0; // change static to non-static

    std::map<std::string, BufferByte*> bufferViewDevPointers;

    // 1. copy all `bufferViews` to device memory
    {
        auto it(scene.bufferViews.begin());
        auto itEnd(scene.bufferViews.end());

        for (; it != itEnd; it++) {
            const std::string key = it->first;
            const tinygltf::BufferView &bufferView = it->second;
            if (bufferView.target == 0) {
                continue; // Unsupported bufferView.
            }

            const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

            BufferByte* dev_bufferView;
            hipMalloc(&dev_bufferView, bufferView.byteLength);
            hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

            checkCUDAError("Set BufferView Device Mem");

            bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

        }
    }



    // 2. for each mesh:
    //		for each primitive:
    //			build device buffer of indices, materail, and each attributes
    //			and store these pointers in a map
    {
        std::map<std::string, glm::mat4> nodeString2Matrix;
        auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

        {
            auto it = rootNodeNamesList.begin();
            auto itEnd = rootNodeNamesList.end();
            for (; it != itEnd; ++it) {
                _traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
            }
        }


        // parse through node to access mesh

        auto itNode = nodeString2Matrix.begin();
        auto itEndNode = nodeString2Matrix.end();
        for (; itNode != itEndNode; ++itNode) {

            const tinygltf::Node & N = scene.nodes.at(itNode->first);
            const glm::mat4 & matrix = itNode->second;
            const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

            auto itMeshName = N.meshes.begin();
            auto itEndMeshName = N.meshes.end();

            for (; itMeshName != itEndMeshName; ++itMeshName) {

                const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

                auto res = sceneInfo.mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveBuffer>>(mesh.name, std::vector<PrimitiveBuffer>()));
                std::vector<PrimitiveBuffer> & primitiveVector = (res.first)->second;

                // for each primitive
                for (size_t i = 0; i < mesh.primitives.size(); i++) {
                    const tinygltf::Primitive &primitive = mesh.primitives[i];

                    if (primitive.indices.empty())
                        return;

                    // TODO: add new attributes for your PrimitiveBuffer when you add new attributes
                    VertexIndex* dev_indices = nullptr;
                    glm::vec3* dev_position = nullptr;
                    glm::vec3* dev_normal = nullptr;
                    glm::vec2* dev_texcoord0 = nullptr;

                    // ----------Indices-------------

                    const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
                    const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
                    BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

                    // assume type is SCALAR for indices
                    int n = 1;
                    int numIndices = indexAccessor.count;
                    int componentTypeByteSize = sizeof(VertexIndex);
                    int byteLength = numIndices * n * componentTypeByteSize;

                    dim3 numThreadsPerBlock(defaultThreadPerBlock);
                    dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    hipMalloc(&dev_indices, byteLength);
                    _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                            numIndices,
                            (BufferByte*)dev_indices,
                            dev_bufferView,
                            n,
                            indexAccessor.byteStride,
                            indexAccessor.byteOffset,
                            componentTypeByteSize);


                    checkCUDAError("Set Index Buffer");


                    // ---------Primitive Info-------

                    // Warning: LINE_STRIP is not supported in tinygltfloader
                    int numPrimitives;
                    PrimitiveType primitiveType;
                    switch (primitive.mode) {
                        case TINYGLTF_MODE_TRIANGLES:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices / 3;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_STRIP:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_FAN:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_LINE:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices / 2;
                            break;
                        case TINYGLTF_MODE_LINE_LOOP:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices + 1;
                            break;
                        case TINYGLTF_MODE_POINTS:
                            primitiveType = PrimitiveType::Point;
                            numPrimitives = numIndices;
                            break;
                        default:
                            // output error
                            break;
                    }


                    // ----------Attributes-------------

                    auto it(primitive.attributes.begin());
                    auto itEnd(primitive.attributes.end());

                    int numVertices = 0;
                    // for each attribute
                    for (; it != itEnd; it++) {
                        const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
                        const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

                        int n = 1;
                        if (accessor.type == TINYGLTF_TYPE_SCALAR) {
                            n = 1;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC2) {
                            n = 2;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC3) {
                            n = 3;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC4) {
                            n = 4;
                        }

                        BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
                        BufferByte ** dev_attribute = nullptr;

                        numVertices = accessor.count;
                        int componentTypeByteSize;

                        // Note: since the type of our attribute array (dev_position) is static (float32)
                        // We assume the glTF model attribute type are 5126(FLOAT) here

                        if (it->first.compare("POSITION") == 0) {
                            componentTypeByteSize = sizeof(glm::vec3) / n;
                            dev_attribute = (BufferByte**)&dev_position;
                        }
                        else if (it->first.compare("NORMAL") == 0) {
                            componentTypeByteSize = sizeof(glm::vec3) / n;
                            dev_attribute = (BufferByte**)&dev_normal;
                        }
                        else if (it->first.compare("TEXCOORD_0") == 0) {
                            componentTypeByteSize = sizeof(glm::vec2) / n;
                            dev_attribute = (BufferByte**)&dev_texcoord0;
                        }

                        // std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

                        dim3 numThreadsPerBlock(defaultThreadPerBlock);
                        dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                        int byteLength = numVertices * n * componentTypeByteSize;
                        hipMalloc(dev_attribute, byteLength);

                        _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                                n * numVertices,
                                *dev_attribute,
                                dev_bufferView,
                                n,
                                accessor.byteStride,
                                accessor.byteOffset,
                                componentTypeByteSize);

                        std::string msg = "Set Attribute Buffer: " + it->first;
                        checkCUDAError(msg.c_str());
                    }

                    // malloc for VertexOut
                    VertexOut* dev_vertexOut;
                    hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
                    checkCUDAError("Malloc VertexOut Buffer");

                    // ----------Materials-------------

                    // You can only worry about this part once you started to
                    // implement textures for your rasterizer
                    MaterialType materialType = Invalid;

                    Tex diffuseTex{nullptr,0,0};
                    Tex specularTex{nullptr,0,0};
                    Tex normalTex{nullptr,0,0};
                    Tex roughnessTex{nullptr,0,0};
                    Tex emissionTex{nullptr,0,0};
                    Tex environmentTex{nullptr,0,0};
                    Tex bakedTex{nullptr,0,0};


                    if (!primitive.material.empty()) {
                        const tinygltf::Material &mat = scene.materials.at(primitive.material);
                        // printf("[Debug] material.name = %s\n", mat.name.c_str());

                        _initTex(scene, mat, "diffuse", diffuseTex);
                        _initTex(scene, mat, "specular", specularTex);
                        _initTex(scene, mat, "normal", normalTex);
                        _initTex(scene, mat, "roughness", roughnessTex);
                        _initTex(scene, mat, "emission", emissionTex);
                        _initTex(scene, mat, "environment", environmentTex);
                        _initTex(scene, mat, "bake", bakedTex);
                    }

                    // Generate material info according to texture;
                    if (diffuseTex.data && specularTex.data && normalTex.data && roughnessTex.data)
                        materialType = PBR;
                    else if (diffuseTex.data)
                        materialType = Tex0;
                    else
                        materialType = Mesh;


                    // ---------Node hierarchy transform--------
                    hipDeviceSynchronize();

                    dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    _nodeMatrixTransform<<<numBlocksNodeTransform, numThreadsPerBlock>>>(
                            numVertices,
                            dev_position,
                            dev_normal,
                            matrix,
                            matrixNormal);

                    checkCUDAError("Node hierarchy transformation");

                    // at the end of the for loop of primitive
                    // push dev pointers to map
                    primitiveVector.push_back(PrimitiveBuffer{
                            primitive.mode,
                            primitiveType,
                            materialType,
                            numPrimitives,
                            numIndices,
                            numVertices,

                            dev_indices,
                            dev_position,
                            dev_normal,

                            dev_texcoord0,
                            {diffuseTex,
                             specularTex,
                             normalTex,
                             roughnessTex,
                             emissionTex,
                             environmentTex,
                             bakedTex},

                            dev_vertexOut	//VertexOut
                    });

                    totalNumPrimitives += numPrimitives;

                } // for each primitive

            } // for each mesh

        } // for each node

    }


    // 3. Malloc for dev_primitives
    {
        sceneInfo.numPrimitives = totalNumPrimitives;
        hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
    }


    // Finally, hipFree raw dev_bufferViews
    {

        std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
        std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());

        //bufferViewDevPointers

        for (; it != itEnd; it++) {
            hipFree(it->second);
        }

        checkCUDAError("Free BufferView Device Mem");
    }

}

